#include "hip/hip_runtime.h"
#include "modernGPU.cuh"
#include "stdio.h"

#include <thrust/scan.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>


#define SHARED_SIZE 512
#define VT 3

void reduce_wrapper(uint numBlocks,
                    uint numThreads,
                    int* result,
                    int* vector,
                    int  vectorSize,
                    int vt){

    k_reduce <<<numBlocks,numThreads>>> (result, vector, vectorSize, vt);

}

__global__ void k_reduce(int* result, int* vector, int vectorSize, int vt){

    int gIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (gIdx >= vectorSize) return;

    int tIdx = threadIdx.x;
    __shared__ int s_vector[SHARED_SIZE];

    //Load values in shared memory
    int partial_sum = 0;
    for (int i= 0; i < VT; i++){

        int global_index = gIdx * VT + i;
        if (global_index < vectorSize)
            partial_sum += vector[global_index];

    }

    s_vector[tIdx] = partial_sum;
    __syncthreads();

    for (int i = blockDim.x; i >=1; i /= 2)
    {
        if (tIdx < i){
            s_vector[tIdx] += s_vector[i + tIdx];
        }
        __syncthreads();
    }

    //store value in global memory
    if (tIdx==0)
        result[blockIdx.x] = s_vector[tIdx];
}

__host__ __device__ uint iDivUp(uint a,
                                uint b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

__host__ __device__ void computeGridSize(uint n,
                                         uint blockSize,
                                         uint &numBlocks,
                                         uint &numThreads)
{
    uint min = blockSize;

    if (min > n)
        min = n;
    numThreads =min;
    numBlocks = iDivUp(n, numThreads);
}

void exclusiveScan_thrust(int *first,
                          int *last,
                          int *result,
                          int init)
{
    thrust::plus<int> binary_op;
    thrust::exclusive_scan(thrust::device,
                           first,
                           last,
                           result,
                           init,
                           binary_op);

}

void exclusiveScan_wrapper(uint numBlocks,
                           uint numThreads,
                           int* result,
                           int* vector,
                           int  vectorSize,
                           int vt){

    int* localScan;
    hipMalloc((void **) &localScan, iDivUp(vectorSize,vt) * sizeof(int));

    int* interBlockScan;
    hipMalloc((void **) &interBlockScan, numBlocks * sizeof(int));

    int* blockSums;
    hipMalloc((void **) &blockSums, numBlocks * sizeof(int));

    //printf("numBlocks %d\n", numBlocks);

    //Perform scans in each CTA, and store the total number in each block in interBlockScan
    k_upsweep <<<numBlocks,numThreads>>> (localScan, blockSums, vector, vectorSize, vt, iDivUp(vectorSize,vt));

    uint numBlocks_exScan = iDivUp(numBlocks, numThreads);
    uint numThreads_exScan;
    //printf("Numblocks_exScan %d\n",numBlocks_exScan);
    computeGridSize(numBlocks,128,numBlocks_exScan,numThreads_exScan);

    if(numBlocks_exScan == 1)
        k_exclusiveScan <<< numBlocks_exScan, numThreads_exScan>>> (interBlockScan, blockSums, numBlocks, 1);
    else
        exclusiveScan_wrapper(numBlocks_exScan,
                              numThreads_exScan,
                              interBlockScan,
                              blockSums,
                              numBlocks,
                              VT);

    //Add to each block the carry-on of its respective block
    k_downsweep <<< numBlocks, numThreads >>> (result, vector, localScan, interBlockScan, vt, iDivUp(vectorSize,vt));

    hipFree(localScan);
    hipFree(interBlockScan);
    hipFree(blockSums);
}

__global__ void k_downsweep(int* result,
                            int* originalArray,
                            int* parallelScans,
                            int* blocksExclusiveScan,
                            int vt,
                            int size)
{

    int gIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (gIdx >= size) return;

    int tIdx = threadIdx.x;
    int carryOn = 0;

    if (blockIdx.x != 0){
        carryOn = blocksExclusiveScan[blockIdx.x];
    }

    //Scan the VT values locally
    int localCarryOn = 0;
    if (tIdx != 0)
        localCarryOn = parallelScans[gIdx];

    int currentSum = 0;

    #pragma unroll
    for (int i=0; i < VT; i++)
    {
        if (tIdx == 0 && i==0)
            currentSum=0;
        else
            currentSum += originalArray[gIdx * VT + i - 1];
       result[gIdx*VT + i] = currentSum + localCarryOn + carryOn;
    }

}

__global__ void k_exclusiveScan(int* result, int*vector, int vectorSize, int vt)
{
    int gIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (gIdx >= vectorSize) return;

    int tIdx = threadIdx.x;
    __shared__ int s_vector[SHARED_SIZE];

    //Load values in shared memory
    int partial_sum = 0;

    #pragma unroll
    for (int i= 0; i < vt; i++){

        int global_index = gIdx * vt + i;
        if (global_index < vectorSize)
            partial_sum += vector[global_index];

    }

    s_vector[tIdx] = partial_sum;
    int first = 0;
    __syncthreads();

    #pragma unroll
    for (int offset = 1; offset < blockDim.x; offset += offset)
    {
        if (tIdx >= offset){
            partial_sum += s_vector[first + tIdx - offset];
        }
        first = blockDim.x - first;
        s_vector[first + tIdx] = partial_sum;
        __syncthreads();
    }

    if (tIdx != 0){
        result[gIdx] = s_vector[tIdx + first - 1];
    }else{
        result[gIdx] = 0;
    }
}



__global__ void k_upsweep(int* result, int* partialSums, int* vector, int vectorSize, int vt, int realSize){

    int gIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (gIdx >= realSize) return;

    int tIdx = threadIdx.x;
    __shared__ int s_vector[SHARED_SIZE];

    //Load values in shared memory
    int partial_sum = 0;

    #pragma unroll
    for (int i= 0; i < VT; i++){

        int global_index = gIdx * VT + i;
        if (global_index < vectorSize && (tIdx + i) != 0)
            partial_sum += vector[global_index - 1];
    }

    s_vector[tIdx] = partial_sum;
    int first = 0;
    __syncthreads();

    #pragma unroll
    for (int offset = 1; offset < blockDim.x; offset += offset)
    {
        if (tIdx >= offset){
            partial_sum += s_vector[first + tIdx - offset];
        }
        first = blockDim.x - first;
        s_vector[first + tIdx] = partial_sum;
        __syncthreads();
    }

    if (tIdx != 0){
        result[gIdx] = s_vector[tIdx + first - 1];
    }else{
        result[gIdx] = 0;
    }

    int lastElem = 0;
    if (blockIdx.x == 0)
        lastElem = vector[blockDim.x * VT-1];
    else if (blockIdx.x == gridDim.x -1)
        lastElem = vector[vectorSize-1];
    else
        lastElem = vector[((blockIdx.x +1) * blockDim.x * VT) -1];

    if (tIdx == 0)
        partialSums[blockIdx.x] = s_vector[blockDim.x + first - 1] + lastElem;

}

